#include "hip/hip_runtime.h"
#include "FixChargePairDebyeHuckel.h"
#include "BoundsGPU.h"
#include "GPUData.h"
#include "GridGPU.h"
#include "State.h"

#include "boost_for_export.h"
#include "cutils_func.h"
#include "EvaluatorWrapper.h"
#include "PairEvaluatorNone.h"
// #include <cmath>

namespace py=boost::python;
using namespace std;

const std::string chargePairDHType = "ChargePairDH";

//Pairwise Debye Huckel 
//force calculation:
//  F=q_i*q_j/(4*PI*eps_0*eps*r_ij)*exp(-r_ij/lambda_D)*[1/r_ij*(1/r_ij + 1/lambda_D)]



//    compute_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), gpd.qs(activeIdx), alpha, r_cut, A, shift, state->boundsGPU, state->devManager.prop.warpSize, 0.5);// state->devManager.prop.warpSize, sigmas.getDevData(), epsilons.getDevData(), numTypes, state->rCut, state->boundsGPU, oneFourStrength);

FixChargePairDH::FixChargePairDH(SHARED(State) state_, string handle_, string groupHandle_) : FixCharge(state_, handle_, groupHandle_, chargePairDHType, true) {
   setParameters(temp, ionic, r_cut);
   canOffloadChargePairCalc = true;
   setEvalWrapper();
};

void FixChargePairDH::setParameters(float temp_, float ionic_, float r_cut_)
{
    temp = temp_;
    ionic = ionic_;
    r_cut = r_cut_;
    
    //Charge of an electron
    double ec = 1.60217653E-19;
    double temp_eps0 = 8.8541878176E-22;
    double na = 6.0221415E23;
    double kb = 1.3806505E-23;
    
    //Calculate the epsilon parameter as a function of temperature and ionic strength
    double temp_eps = (249.4 - 0.788 * temp + 7.20E-4 * temp * temp);
    temp_eps *= (1.000 - 0.2551 * ionic + 5.151E-2 * ionic * ionic - 6.889E-3 * ionic * ionic * ionic);

    //Calculate the debye length from the given temperature and ionic strength calculations
    double lambda = sqrt(temp_eps * temp_eps0 * kb * temp * 1.0E27 / (2.0f * na * ec * ec * ionic)); 
    //We store the inverse as that is the only form that is used by the Evaluator
    lambdai = 1.0 / lambda;
    //Convert epsilon to kcal/mol from kJ/mol
    epsi = 1.0 / (temp_eps);  
    //printf("Epsi is %f, debye length is %f\n", 1.0/epsi, 1.0/lambdai);
}

void FixChargePairDH::compute(int virialMode) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    evalWrap->compute(nAtoms, nPerRingPoly, gpd.xs(activeIdx), gpd.fs(activeIdx),
                  neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
                  state->devManager.prop.warpSize, nullptr, 0, state->boundsGPU,
                  neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.virials.d_data.data(), gpd.qs(activeIdx), r_cut, virialMode,  nThreadPerBlock(), nThreadPerAtom());
  //  compute_charge_pair_DH_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborIdxs, grid.neighborlist.tex, gpd.qs(activeIdx), alpha,r_cut, A,shift, state->boundsGPU, 0.5);
}

std::vector<float> FixChargePairDH::getRCuts() {
    std::vector<float> vals(1, state->rCut);
    return vals;
}

void FixChargePairDH::setEvalWrapper() {
    if (evalWrapperMode == "offload") {
        if (hasOffloadedChargePairCalc) {
            evalWrap = pickEvaluator<EvaluatorNone, 1, false>(EvaluatorNone(), nullptr); //nParams arg is 1 rather than zero b/c can't have zero sized argument on device
        } else {
            evalWrap = pickEvaluator<EvaluatorNone, 1, false>(EvaluatorNone(), this);
        }
    } else if (evalWrapperMode == "self") {
        evalWrap = pickEvaluator<EvaluatorNone, 1, false>(EvaluatorNone(), this);
    }

}

ChargeEvaluatorDH FixChargePairDH::generateEvaluator() {
    return ChargeEvaluatorDH(lambdai, epsi, state->units.qqr_to_eng);
}

void export_FixChargePairDH() {
    py::class_<FixChargePairDH, SHARED(FixChargePairDH), boost::python::bases<FixCharge> > (
        "FixChargePairDH",
        py::init<SHARED(State), string, string> (
            py::args("state", "handle", "groupHandle"))
    )
    .def("setParameters", &FixChargePairDH::setParameters,
            (py::arg("temp"), py::arg("ionic"), py::arg("r_cut"))
        )
    ;
}

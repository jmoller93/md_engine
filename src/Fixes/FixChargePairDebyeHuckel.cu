#include "hip/hip_runtime.h"
#include "FixChargePairDebyeHuckel.h"
#include "BoundsGPU.h"
#include "GPUData.h"
#include "GridGPU.h"
#include "State.h"

#include "boost_for_export.h"
#include "cutils_func.h"
// #include <cmath>

namespace py=boost::python;
using namespace std;

const std::string chargePairDHType = "ChargePairDH";

//Pairwise Debye Huckel 
//force calculation:
//  F=q_i*q_j/(4*PI*eps_0*eps*r_ij)*exp(-r_ij/lambda_D)*[1/r_ij*(1/r_ij + 1/lambda_D)]



//    compute_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), gpd.qs(activeIdx), alpha, r_cut, A, shift, state->boundsGPU, state->devManager.prop.warpSize, 0.5);// state->devManager.prop.warpSize, sigmas.getDevData(), epsilons.getDevData(), numTypes, state->rCut, state->boundsGPU, oneFourStrength);
__global__ void compute_charge_pair_DH_cu(int nAtoms, float4 *xs, float4 *fs, uint16_t *neighborCounts, uint *neighborlist, uint32_t *cumulSumMaxPerBlock, float *qs, float lambdai, float epsi, BoundsGPU bounds, int warpSize, float onetwoStr, float onethreeStr, float onefourStr) {

    float multipliers[4] = {1, onetwoStr, onethreeStr, onefourStr};
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole);

        float3 forceSum = make_float3(0, 0, 0);
        float qi = qs[idx];//tex2D<float>(qs, XIDX(idx, sizeof(float)), YIDX(idx, sizeof(float)));

        //printf("start, end %d %d\n", start, end);
        int baseIdx = baseNeighlistIdx(cumulSumMaxPerBlock, warpSize);
        int numNeigh = neighborCounts[idx];
        for (int i=0; i<numNeigh; i++) {
            int nlistIdx = baseIdx + warpSize * i;
            uint otherIdxRaw = neighborlist[nlistIdx];
            uint neighDist = otherIdxRaw >> 30;
            uint otherIdx = otherIdxRaw & EXCL_MASK;
            float3 otherPos = make_float3(xs[otherIdx]);
            //then wrap and compute forces!
            float3 dr = bounds.minImage(pos - otherPos);
            float lenSqr = lengthSqr(dr);
            //   printf("dist is %f %f %f\n", dr.x, dr.y, dr.z);
            float multiplier = multipliers[neighDist];
            float len=sqrtf(lenSqr);
            float qj = qs[otherIdx];

            float rinv = 1.0f/len;
            float forceScalar = qi*qj*epsi*expf(-len*lambdai)*(rinv*(rinv+lambdai)) * multiplier;
    
            float3 forceVec = dr * forceScalar;
            forceSum += forceVec;

        }   
        fs[idx] += forceSum; //operator for float4 + float3

    }

}
FixChargePairDH::FixChargePairDH(SHARED(State) state_, string handle_, string groupHandle_) : FixCharge(state_, handle_, groupHandle_, chargePairDHType, true) {
   setParameters(temp, ionic);
};

void FixChargePairDH::setParameters(float temp_, float ionic_)
{
    temp = temp_;
    ionic = ionic_;
    double ec = 1.60217653E-19;
    double temp_eps0 = 8.8541878176E-22;
    double temp_eps = ((249.4 - 0.788 * temp + 7.20E-4 * temp * temp) * 
                     (1.000 - 2.551 * ionic + 5.151E-2 * ionic * ionic -
                      6.889E-3 * ionic * ionic * ionic) * temp_eps0
                     )
                    ;
    double lambda = sqrt(temp_eps * 1.3806505E-23 * 300.0 * 1.0E30 / (2.0 * 6.0221415E23 * ec * ec * ionic)); 
    lambdai = 1.0 / lambda;
    epsi = 1.0 / (4.0 * M_PI * temp_eps);  
}

void FixChargePairDH::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    compute_charge_pair_DH_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), gpd.qs(activeIdx), lambdai, epsi, state->boundsGPU, state->devManager.prop.warpSize, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2]);// state->devManager.prop.warpSize, sigmas.getDevData(), epsilons.getDevData(), numTypes, state->rCut, state->boundsGPU, oneFourStrength);
  //  compute_charge_pair_DH_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborIdxs, grid.neighborlist.tex, gpd.qs(activeIdx), alpha,r_cut, A,shift, state->boundsGPU, 0.5);


}


void export_FixChargePairDH() {
    py::class_<FixChargePairDH, SHARED(FixChargePairDH), boost::python::bases<FixCharge> > (
        "FixChargePairDH",
        py::init<SHARED(State), string, string> (
            py::args("state", "handle", "groupHandle"))
    )
    .def("setParameters", &FixChargePairDH::setParameters,
            (py::arg("temp"), py::arg("ionic"))
        )
    ;
}

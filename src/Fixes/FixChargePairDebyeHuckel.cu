#include "hip/hip_runtime.h"
#include "FixChargePairDebyeHuckel.h"
#include "BoundsGPU.h"
#include "GPUData.h"
#include "GridGPU.h"
#include "State.h"

#include "boost_for_export.h"
#include "cutils_func.h"
// #include <cmath>

namespace py=boost::python;
using namespace std;

const std::string chargePairDHType = "ChargePairDH";

//Pairwise Debye Huckel 
//force calculation:
//  F=q_i*q_j/(4*PI*eps_0*eps*r_ij)*exp(-r_ij/lambda_D)*[1/r_ij*(1/r_ij + 1/lambda_D)]



//    compute_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), gpd.qs(activeIdx), alpha, r_cut, A, shift, state->boundsGPU, state->devManager.prop.warpSize, 0.5);// state->devManager.prop.warpSize, sigmas.getDevData(), epsilons.getDevData(), numTypes, state->rCut, state->boundsGPU, oneFourStrength);
__global__ void compute_charge_pair_DH_cu(int nAtoms, float4 *xs, float4 *fs, uint16_t *neighborCounts, uint *neighborlist, uint32_t *cumulSumMaxPerBlock, float *qs, float lambdai, float epsi, float qqr_to_eng,  BoundsGPU bounds, int warpSize, float onetwoStr, float onethreeStr, float onefourStr) {

    float multipliers[4] = {1, onetwoStr, onethreeStr, onefourStr};
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole);

        float3 forceSum = make_float3(0, 0, 0);
        float qi = qs[idx];//tex2D<float>(qs, XIDX(idx, sizeof(float)), YIDX(idx, sizeof(float)));

        //printf("start, end %d %d\n", start, end);
        int baseIdx = baseNeighlistIdx(cumulSumMaxPerBlock, warpSize);
        int numNeigh = neighborCounts[idx];
        for (int i=0; i<numNeigh; i++) {
            int nlistIdx = baseIdx + warpSize * i;
            uint otherIdxRaw = neighborlist[nlistIdx];
            uint neighDist = otherIdxRaw >> 30;
            uint otherIdx = otherIdxRaw & EXCL_MASK;
            float3 otherPos = make_float3(xs[otherIdx]);
            //then wrap and compute forces!
            float3 dr = bounds.minImage(pos - otherPos);
            float lenSqr = lengthSqr(dr);
            //   printf("dist is %f %f %f\n", dr.x, dr.y, dr.z);
            float multiplier = multipliers[neighDist];
            float len=sqrtf(lenSqr);
            float qj = qs[otherIdx];

            float rinv = 1.0f/len;
            float forceScalar = qi*qj*epsi*rinv*expf(-len*lambdai)*(rinv*(rinv+lambdai)) * multiplier;
    
            float3 forceVec = dr * forceScalar;
            forceSum += forceVec;

        }   
        fs[idx] += forceSum; //operator for float4 + float3

    }

}
FixChargePairDH::FixChargePairDH(SHARED(State) state_, string handle_, string groupHandle_) : FixCharge(state_, handle_, groupHandle_, chargePairDHType, true) {
   setParameters(temp, ionic);
   canOffloadChargePairCalc = true;
};

void FixChargePairDH::setParameters(float temp_, float ionic_)
{
    temp = temp_;
    ionic = ionic_;
    
    //Charge of an electron
    double ec = 1.60217653E-19;
    double temp_eps0 = 8.8541878176E-22;
    
    //Calculate the epsilon parameter as a function of temperature and ionic strength
    double temp_eps = ((249.4 - 0.788 * temp + 7.20E-4 * temp * temp) * 
                     (1.000 - 0.2551 * ionic + 5.151E-2 * ionic * ionic -
                      6.889E-3 * ionic * ionic * ionic)
                     )
                    ;

    //Calculate the debye length from the given temperature and ionic strength calculations
    double lambda = sqrt(temp_eps * temp_eps0 * 1.3806505E-23 * temp * 1.0E30 / (2.0f * 6.0221415E23 * ec * ec * ionic)); 
    //We store the inverse as that is the only form that is used by the Evaluator
    lambdai = 1.0 / lambda;
    //Convert epsilon to kcal/mol from kJ/mol
    epsi = 1.0 / (temp_eps);  
    printf("Epsi is %f, debye length is %f\n", 1.0/epsi, 1.0/lambdai);
}

void FixChargePairDH::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    compute_charge_pair_DH_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), gpd.qs(activeIdx), lambdai, epsi, state->units.qqr_to_eng, state->boundsGPU, state->devManager.prop.warpSize, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2]);// state->devManager.prop.warpSize, sigmas.getDevData(), epsilons.getDevData(), numTypes, state->rCut, state->boundsGPU, oneFourStrength);
  //  compute_charge_pair_DH_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neighborIdxs, grid.neighborlist.tex, gpd.qs(activeIdx), alpha,r_cut, A,shift, state->boundsGPU, 0.5);


}


std::vector<float> FixChargePairDH::getRCuts() {
    std::vector<float> vals(1, state->rCut);
    return vals;
}

void export_FixChargePairDH() {
    py::class_<FixChargePairDH, SHARED(FixChargePairDH), boost::python::bases<FixCharge> > (
        "FixChargePairDH",
        py::init<SHARED(State), string, string> (
            py::args("state", "handle", "groupHandle"))
    )
    .def("setParameters", &FixChargePairDH::setParameters,
            (py::arg("temp"), py::arg("ionic"))
        )
    ;
}
